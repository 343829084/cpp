
#include <hip/hip_runtime.h>
#include<stdio.h>
#include <stdlib.h>
#include <unistd.h>
//#include <conio.h>
#ifndef __host__
    #define __host__
    #define __device__
    #define __global__
    #define __CUDACC__
    #define __host__
    #define __device__
    #define __global__
    #define __forceinline__
    #define __shared__

    inline void __syncthreads()
    {}

    inline void __threadfence_block()
    {}

    template<class T>
    inline T __clz(const T val)
    { return val; }

    struct __cuda_fake_struct
    {
        int x;
        int y;
        int z;
    };
    extern __cuda_fake_struct blockDim;
    extern __cuda_fake_struct threadIdx;
    extern __cuda_fake_struct blockIdx;
    extern __cuda_fake_struct gridDim;
    #define void cudaMemcpy(void *dst, const void *src, size_t count, enum cudaMemcpyKind kind)
    #define  void cudaMalloc(void **devPtr, size_t size)
    #define warpSize
    #define cudaFree void cudaFree(void *devPtr)

    #define cudaMemcpyDeviceToHost
    //#define  dim3 struct dim3{int x;int y;int z ;}dim3;
    extern __cuda_fake_struct dim3;
#endif

__global__ void what_is_my_id_2D_A(unsigned int *const block_x,
                                   unsigned int *const block_y,
                                   unsigned int *const thread,
                                   unsigned int *const calc_thread,
                                   unsigned int *const x_thread,
                                   unsigned int *const y_thread,
                                   unsigned int *const grid_dimx,
                                   unsigned int *const block_dimx,
                                   unsigned int *const grid_dimy,
                                   unsigned int *const block_dimy
)
{
    /*
     *threadIdx.x; 某一具体线程在线程块x方向上的位置,x方向的偏移。
     *threadIdx.y;
     * blockIdx.x:线程块在线程网格的x方向的索引。
     * blockIdx.y;线程块在线程网格的y方向的索引。
     *
     *blockDim.x;线程块的宽度,x方向的【线程】数量
     *blockDim.y;线程块的高度,y方向的【线程】数量
     * gridDim.x; 线程网格的宽度,x维度上【线程块】的数量
     * gridDim.y; 线程网格的高度,y维度上【线程块】的数量
     * */

    const unsigned int idx=(blockIdx.x*blockDim.x)+threadIdx.x;
    const unsigned  int idy=(blockIdx.y*blockDim.y)*threadIdx.y;

    //找出当前的行索引，乘以每一行的线程数，最后加上在x方向上的偏移。便是相对于整个线程网格的绝对线程索引。
    const unsigned int thread_idx=((gridDim.x*blockDim.x)*idy)+idx;
    block_x[thread_idx]=blockIdx.x;//线程在线程块的x方向的偏移
    block_y[thread_idx]=blockIdx.y;//线程在线程块的y方向的偏移

    thread[thread_idx]=threadIdx.x;//线程块x维度上的线程索引。
    calc_thread[thread_idx]=thread_idx;
    x_thread[thread_idx]=idx;
    y_thread[thread_idx]=idy;
    grid_dimx[thread_idx]=gridDim.x;   //线程网格x维度上线程块的数量
    block_dimx[thread_idx]=blockDim.x; //一个线程块x维度上的线程数量

    grid_dimy[thread_idx]=gridDim.y;   //线程网格y维度上【线程块】的数量
    block_dimy[thread_idx]=blockDim.y;//一个线程块y维度上的【线程】数量
};

#define ARRAY_SIZE_X 32
#define ARRAY_SIZE_Y 16

#define ARRAY_SIZE_IN_BYTES ((ARRAY_SIZE_X)*(ARRAY_SIZE_Y)*sizeof(unsigned int ))

unsigned int cpu_block_x[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_y[ARRAY_SIZE_Y][ARRAY_SIZE_X];

unsigned int cpu_thread[ARRAY_SIZE_Y][ARRAY_SIZE_X];

unsigned int cpu_warp[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_calc_thread[ARRAY_SIZE_Y][ARRAY_SIZE_X];


unsigned int cpu_xthread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_ythread[ARRAY_SIZE_Y][ARRAY_SIZE_X];


unsigned int cpu_grid_dimx[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_dimx[ARRAY_SIZE_Y][ARRAY_SIZE_X];

unsigned int cpu_grid_dimy[ARRAY_SIZE_Y][ARRAY_SIZE_X];

unsigned int cpu_block_dimy[ARRAY_SIZE_Y][ARRAY_SIZE_X];
/*
 * 写一个简短的内核程序来输出线程块，线程，线程束，线程全局标号。
 * */
int main()
{

    const dim3 thread_rect(32,4); //32*4=128个线程
    const dim3 blocks_rect(1,4);  //长方形，每个线程块的排列方式。

    //或者:
    const dim3 threads_square(16*8);//128个线程
    const dim3 blocks_square(2,2);


    unsigned int *gpu_block_x;
    unsigned int *gpu_block_y;
    unsigned int *gpu_thread;
    unsigned int *gpu_warp;
    unsigned int *gpu_calc_thread;
    unsigned int *gpu_xthread;
    unsigned int *gpu_ythread;

    unsigned int *gpu_grid_dimx;
    unsigned int *gpu_block_dimx;

    unsigned int *gpu_grid_dimy;
    unsigned int *gpu_block_dimy;

    hipMalloc((void**)&gpu_block_x,ARRAY_SIZE_IN_BYTES);
    hipMalloc((void**)&gpu_block_y,ARRAY_SIZE_IN_BYTES);
    hipMalloc((void**)&gpu_thread,ARRAY_SIZE_IN_BYTES);
    hipMalloc((void**)&gpu_calc_thread,ARRAY_SIZE_IN_BYTES);
    hipMalloc((void**)&gpu_xthread,ARRAY_SIZE_IN_BYTES);
    hipMalloc((void**)&gpu_ythread,ARRAY_SIZE_IN_BYTES);
    hipMalloc((void**)&gpu_grid_dimx,ARRAY_SIZE_IN_BYTES);
    hipMalloc((void**)&gpu_block_dimx,ARRAY_SIZE_IN_BYTES);
    hipMalloc((void**)&gpu_grid_dimy,ARRAY_SIZE_IN_BYTES);
    hipMalloc((void**)&gpu_block_dimy,ARRAY_SIZE_IN_BYTES);

    for (int i = 0; i < 1; ++i) //kernel:0-1
    {
        switch (i)
        {
            case 0:
            {
                what_is_my_id_2D_A<<<blocks_rect,thread_rect>>>(gpu_block_x,
                        gpu_block_y,
                        gpu_thread,
                        gpu_calc_thread,
                        gpu_xthread,
                        gpu_ythread,
                        gpu_grid_dimx,
                        gpu_block_dimx,
                        gpu_grid_dimy,
                        gpu_block_dimy
                );
            }
                break;
            case 1:
            {
                what_is_my_id_2D_A<<<blocks_rect,thread_rect>>>(gpu_block_x,
                        gpu_block_y,
                        gpu_thread,
                        gpu_calc_thread,
                        gpu_xthread,
                        gpu_ythread,
                        gpu_grid_dimx,
                        gpu_block_dimx,
                        gpu_grid_dimy,
                        gpu_block_dimy
                );
            }
            default:
                exit(1);
                break;
        }

        hipMemcpy(cpu_block_x,gpu_block_x,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
        hipMemcpy(cpu_block_y,gpu_block_y,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
        hipMemcpy(cpu_thread,gpu_thread,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
        hipMemcpy(cpu_calc_thread,gpu_calc_thread,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
        hipMemcpy(cpu_xthread,gpu_xthread,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
        hipMemcpy(cpu_ythread,gpu_ythread,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
        hipMemcpy(cpu_grid_dimx,gpu_grid_dimx,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
        hipMemcpy(cpu_block_dimx,gpu_block_dimx,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
        hipMemcpy(cpu_grid_dimy,gpu_grid_dimy,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
        hipMemcpy(cpu_block_dimy,gpu_block_dimy,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
        printf("\nkernel %d\n",i);
        for (int y = 0; y < ARRAY_SIZE_Y; ++y)
        {
            for (int x = 0; x < ARRAY_SIZE_X; ++x)
            {
                printf("CT:%2u,BKX:%1u,TID：%2u,YID：%2u,XTID:%2u,GDX:%1u,BDX:%1u,GDY:%1u,BDY:%1u\n",
                       cpu_calc_thread[y][x],
                       cpu_block_x[y][x],
                       cpu_block_y[y][x],
                       cpu_thread[y][x],
                       cpu_ythread[y][x],
                       cpu_xthread[y][x],
                       cpu_grid_dimx[y][x],
                       cpu_block_dimx[y][x],
                       cpu_grid_dimy[y][x],
                       cpu_block_dimy[y][x]
                );
            }
        }
        sleep(1);
    }

}