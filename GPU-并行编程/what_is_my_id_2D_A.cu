
#include <hip/hip_runtime.h>
#include<stdio.h>
#include <stdlib.h>
#include <unistd.h>
//#include <conio.h>
#ifndef __host__
    #define __host__
    #define __device__
    #define __global__
    #define __CUDACC__
    #define __host__
    #define __device__
    #define __global__
    #define __forceinline__
    #define __shared__

    inline void __syncthreads()
    {}

    inline void __threadfence_block()
    {}

    template<class T>
    inline T __clz(const T val)
    { return val; }

    struct __cuda_fake_struct
    {
        int x;
        int y;
        int z;
    };
    extern __cuda_fake_struct blockDim;
    extern __cuda_fake_struct threadIdx;
    extern __cuda_fake_struct blockIdx;
    extern __cuda_fake_struct gridDim;
    #define cudaMemcpy
    #define  cudaMalloc
    #define warpSize
    #define cudaFree

    #define cudaMemcpyDeviceToHost
    //#define  dim3 struct dim3{int x;int y;int z ;}dim3;
    extern __cuda_fake_struct dim3;
#endif

__global__ void what_is_my_id_2D_A(unsigned int *const block_x,
                                   unsigned int *const block_y,
                                   unsigned int *const thread,
                                   unsigned int *calc_thread,
                                   unsigned int *const x_thread,
                                   unsigned int *const y_thread,
                                   unsigned int *const grid_dimx,
                                   unsigned int *const block_dimx,
                                   unsigned int *const grid_dimy,
                                   unsigned int *const block_dimy
)
{
    const unsigned int idx=(blockIdx.x*blockDim.x)+threadIdx.x;
    const unsigned  int idy=(blockIdx.y*blockDim.y)*threadIdx.y;
    const unsigned int thread_idx=((gridDim.x*blockDim.x)*idy)+idx;
    block_x[thread_idx]=blockIdx.x;
    block_y[thread_idx]=blockIdx.y;

    thread[thread_idx]=threadIdx.x;
    calc_thread[thread_idx]=thread_idx;
    x_thread[thread_idx]=idx;
    y_thread[thread_idx]=idy;
    grid_dimx[thread_idx]=gridDim.x;
    block_dimx[thread_idx]=blockDim.x;

    grid_dimy[thread_idx]=gridDim.y;
    block_dimy[thread_idx]=blockDim.y;
};

#define ARRAY_SIZE_X 32
#define ARRAY_SIZE_Y 16

#define ARRAY_SIZE_IN_BYTES ((ARRAY_SIZE_X)*(ARRAY_SIZE_Y)*sizeof(unsigned int ))

unsigned int cpu_block_x[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_y[ARRAY_SIZE_Y][ARRAY_SIZE_X];

unsigned int cpu_thread[ARRAY_SIZE_Y][ARRAY_SIZE_X];

unsigned int cpu_warp[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_calc_thread[ARRAY_SIZE_Y][ARRAY_SIZE_X];


unsigned int cpu_xthread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_ythread[ARRAY_SIZE_Y][ARRAY_SIZE_X];


unsigned int cpu_grid_dimx[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_dimx[ARRAY_SIZE_Y][ARRAY_SIZE_X];

unsigned int cpu_grid_dimy[ARRAY_SIZE_Y][ARRAY_SIZE_X];

unsigned int cpu_block_dimy[ARRAY_SIZE_Y][ARRAY_SIZE_X];
/*
 * 写一个简短的内核程序来输出线程块，线程，线程束，线程全局标号。
 * */
int main()
{

    const dim3 thread_rect(32,4);
    const dim3 blocks_rect(1,4);

    const dim3 threads_square(16*8);
    const dim3 blocks_square(2,2);


    unsigned int *gpu_block_x;
    unsigned int *gpu_block_y;
    unsigned int *gpu_thread;
    unsigned int *gpu_warp;
    unsigned int *gpu_calc_thread;
    unsigned int *gpu_xthread;
    unsigned int *gpu_ythread;

    unsigned int *gpu_grid_dimx;
    unsigned int *gpu_block_dimx;

    unsigned int *gpu_grid_dimy;
    unsigned int *gpu_block_dimy;

    hipMalloc((void**)&gpu_block_x,ARRAY_SIZE_IN_BYTES);
    hipMalloc((void**)&gpu_block_y,ARRAY_SIZE_IN_BYTES);
    hipMalloc((void**)&gpu_thread,ARRAY_SIZE_IN_BYTES);
    hipMalloc((void**)&gpu_calc_thread,ARRAY_SIZE_IN_BYTES);
    hipMalloc((void**)&gpu_xthread,ARRAY_SIZE_IN_BYTES);
    hipMalloc((void**)&gpu_ythread,ARRAY_SIZE_IN_BYTES);
    hipMalloc((void**)&gpu_grid_dimx,ARRAY_SIZE_IN_BYTES);
    hipMalloc((void**)&gpu_block_dimx,ARRAY_SIZE_IN_BYTES);
    hipMalloc((void**)&gpu_grid_dimy,ARRAY_SIZE_IN_BYTES);
    hipMalloc((void**)&gpu_block_dimy,ARRAY_SIZE_IN_BYTES);

    for (int i = 0; i < 1; ++i) //kernel:0-1
    {
        switch (i)
        {
            case 0:
            {
                what_is_my_id_2D_A<<<blocks_rect,thread_rect>>>(gpu_block_x,
                        gpu_block_y,
                        gpu_thread,
                        gpu_calc_thread,
                        gpu_xthread,
                        gpu_ythread,
                        gpu_grid_dimx,
                        gpu_block_dimx,
                        gpu_grid_dimy,
                        gpu_block_dimy
                );
            }
                break;
            case 1:
            {
                what_is_my_id_2D_A<<<blocks_rect,thread_rect>>>(gpu_block_x,
                        gpu_block_y,
                        gpu_thread,
                        gpu_calc_thread,
                        gpu_xthread,
                        gpu_ythread,
                        gpu_grid_dimx,
                        gpu_block_dimx,
                        gpu_grid_dimy,
                        gpu_block_dimy
                );
            }
            default:
                exit(1);
                break;
        }

        hipMemcpy(cpu_block_x,gpu_block_x,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
        hipMemcpy(cpu_block_y,gpu_block_y,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
        hipMemcpy(cpu_thread,gpu_thread,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
        hipMemcpy(cpu_calc_thread,gpu_calc_thread,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
        hipMemcpy(cpu_xthread,gpu_xthread,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
        hipMemcpy(cpu_ythread,gpu_ythread,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
        hipMemcpy(cpu_grid_dimx,gpu_grid_dimx,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
        hipMemcpy(cpu_block_dimx,gpu_block_dimx,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
        hipMemcpy(cpu_grid_dimy,gpu_grid_dimy,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
        hipMemcpy(cpu_block_dimy,gpu_block_dimy,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
        printf("\nkernel %d\n",i);
        for (int y = 0; y < ARRAY_SIZE_Y; ++y)
        {
            for (int x = 0; x < ARRAY_SIZE_X; ++x)
            {
                printf("CT:%2u,BKX:%1u,TID：%2u,YID：%2u,XTID:%2u,GDX:%1u,BDX:%1u,GDY:%1u,BDY:%1u\n",
                       cpu_calc_thread[y][x],
                       cpu_block_x[y][x],
                       cpu_block_y[y][x],
                       cpu_thread[y][x],
                       cpu_ythread[y][x],
                       cpu_xthread[y][x],
                       cpu_grid_dimx[y][x],
                       cpu_block_dimx[y][x],
                       cpu_grid_dimy[y][x],
                       cpu_block_dimy[y][x]
                );
            }
        }
        sleep(1);
    }

}