#include "hip/hip_runtime.h"
#include<stdio.h>
#include <stdlib.h>
//#include <conio.h>
#ifndef __host__
    #define __host__
    #define __device__
    #define __global__
    #define __HIPCC__
    #define __host__
    #define __device__
    #define __global__
    #define __forceinline__
    #define __shared__

    inline void __syncthreads()
    {}

    inline void __threadfence_block()
    {}

    template<class T>
    inline T __clz(const T val)
    { return val; }

    struct __cuda_fake_struct
    {
        int x;
    };
    extern __cuda_fake_struct blockDim;
    extern __cuda_fake_struct threadIdx;
    extern __cuda_fake_struct blockIdx;
    #define hipMemcpy
    #define  hipMalloc
    #define warpSize
    #define hipFree

    #define hipMemcpyDeviceToHost
#endif

__global__ void what_is_my_id(unsigned int *const block, unsigned int *const thread,
                              unsigned int *const wrap, unsigned int *calc_thread)
{
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    block[thread_idx] = blockIdx.x;
    thread[thread_idx] = threadIdx.x;
    wrap[thread_idx] = threadIdx.x / warpSize;
    calc_thread[thread_idx] = thread_idx;
};

#define ARRAY_SIZE 128
#define ARRAY_SIZE_IN_BYTES (sizeof(unsigned int )* (ARRAY_SIZE))
unsigned int cpu_block[ARRAY_SIZE];
unsigned int cpu_thread[ARRAY_SIZE];
unsigned int cpu_warp[ARRAY_SIZE];
unsigned int cpu_calc_thread[ARRAY_SIZE];
/*
 * 写一个简短的内核程序来输出线程块，线程，线程束，线程全局标号。
 * */
int main()
{
    const unsigned int num_blocks = 2;//线程块
    const unsigned int num_threads = 64;//每块的线程数
    char ch;
    unsigned int *gpu_block;
    unsigned int *gpu_thread;
    unsigned int *gpu_warp;
    unsigned int *gpu_calc_thread;
    unsigned int i;
    hipMalloc((void **) &gpu_block, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **) &gpu_thread, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **) &gpu_warp, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **) &gpu_calc_thread, ARRAY_SIZE_IN_BYTES);

    what_is_my_id << < num_blocks, num_threads >> > (gpu_block, gpu_thread, gpu_warp, gpu_calc_thread);
    hipMemcpy(cpu_block, gpu_block, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);

    hipMemcpy(cpu_thread, gpu_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);

    hipMemcpy(cpu_warp, gpu_warp, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(cpu_calc_thread, gpu_calc_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);

    hipFree(gpu_block);
    hipFree(gpu_thread);
    hipFree(gpu_warp);
    hipFree(gpu_calc_thread);

    for (int i = 0; i < ARRAY_SIZE; ++i)
    {
        printf("cal_thread: %3d, block: %2u,wrap：%2u,thread:%3u\n",
               cpu_calc_thread[i],cpu_block[i],cpu_warp[i],cpu_thread[i]);
    }
    /*
     * 1，线程块按照编号紧密排列，
     * 2，线程索引是0-127
     * 3，一共有2块线程，
     * 4，每个线程块包含64个线程
     * 5，每个线程块的内部索引是0-63
     * */

}